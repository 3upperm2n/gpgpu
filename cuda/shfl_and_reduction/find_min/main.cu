#include "hip/hip_runtime.h"
/*
	MiroBenchmark for SHFL instruction
	Reduction
*/

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

inline int BLK(int number, int blksize)
{
	return (number + blksize - 1) / blksize;
}

void print_2d_array(float* data, int rows, int cols, const char* msg)
{
	// needed for read from unified memory
	hipDeviceSynchronize();

	if(msg != NULL) {
		printf("\n%s:\n", msg);
	}

	for(int i=0; i<rows; i++) {                                                 
		for(int j=0; j<cols; j++) {                                             
			printf("%10.6f ", data[i * cols + j]);                               
		}                                                                       
		printf("\n");                                                           
	}
	printf("rows : %d \t cols : %d\n", rows, cols);
}

/*
static unsigned int iDivUp(unsigned int dividend, unsigned int divisor)
{
    return ((dividend % divisor) == 0) ?
           (dividend / divisor) :
           (dividend / divisor + 1);
}
*/

__global__ void reduction_min_v1_part1 (float *data, float *partial)
{
	uint gx = threadIdx.x + __umul24(blockDim.x, blockIdx.x);

	int lane_id = threadIdx.x & 0x1F;

	float value = data[gx];

	// sum up the 32 threads
	for (int i=16; i>0; i>>=1 ) {
		value = fminf( value, __shfl_down(value, i, 32) );
	}

	if( lane_id == 0){
		partial[blockIdx.x] = value;
	}
}

__global__ void reduction_min_v1_part2 (float *partialdata, const int cols, float *result)
{
	uint gx = threadIdx.x + __umul24(blockDim.x, blockIdx.x);

	float min_data = partialdata[0];
	for(int i=1; i<cols; i++) {
		if(min_data > partialdata[i]) 
			min_data = partialdata[i];
	}
	result[0] = min_data;
}

void gpu_min_test1()
{
	printf("\n\nparallel reduction using shuffle - min op:\n\n");

	// allocation
	int N = 96;
	size_t bytes = N * sizeof(float);
	float *data;
	hipMallocManaged((void**)&data, bytes);

	float *result;
	hipMallocManaged((void**)&result, sizeof(float));

	// init
	for(int i=0; i<N; i++) {
		data[i] = (float) (N - i);
		//printf("%d : %f\n", i, data[i]);
	}


	dim3 Blkdim = dim3(32, 1, 1);
	dim3 Grddim = dim3(BLK(N, 32), 1, 1);

	float *partialdata;
	hipMallocManaged((void**)&partialdata, sizeof(float) * Grddim.x);

	reduction_min_v1_part1 <<< Grddim, Blkdim >>> (data, partialdata);

	print_2d_array(partialdata, Grddim.x, 1, "partial data");

	// gather
	reduction_min_v1_part2 <<< 1, 1 >>> (partialdata, Grddim.x ,result);


	print_2d_array(result, 1, 1, "result");

	// release
	hipHostFree(data);
	hipHostFree(partialdata);
	hipHostFree(result);
	//hipFree(d_partialsum);
}


__global__ void reduction_min_v2 (float *data, const int warp_num, float *result)
{
	extern __shared__ float sdata[];

	uint gx = threadIdx.x + __umul24(blockDim.x, blockIdx.x);

	//int lane_id = threadIdx.x & 0x1F;
	uint lane_id;
	asm("mov.u32 %0,%laneid;" : "=r"(lane_id));

	/// warning:
	///				warp id is not exact in order
	//uint warp_id;
	//asm("mov.u32 %0,%warpid;" : "=r"(warp_id));

	uint warp_id;
	warp_id = threadIdx.x >> 5;

	// printf("thread: %d, lane_id: %d, warp_id: %d\n", gx, lane_id, warp_id);

	float value = data[gx];

	// sum up the 32 threads
	for (int i=16; i>0; i>>=1 ) {
		value = fminf( value, __shfl_down(value, i, 32) );
	}

	if(lane_id == 0) {
		//partial[blockIdx.x] = value;
		sdata[warp_id] = value;
	}

	__syncthreads();

	if(gx == 0)
	{
		float min_data = sdata[0];
		
		/// notes : use define to unroll each case
		for(int i=1; i<warp_num; i++) {
			if(min_data > sdata[i]) 
				min_data = sdata[i];
		}

		result[blockIdx.x] = min_data;
	}
}


void gpu_min_test2()
{
	printf("\n\nparallel reduction using shuffle - min op:  using 1 block \n\n");

	// allocation
	int N = 96;
	size_t bytes = N * sizeof(float);
	float *data;
	hipMallocManaged((void**)&data, bytes);

	float *result;
	hipMallocManaged((void**)&result, sizeof(float));

	// init
	for(int i=0; i<N; i++) {
		data[i] = (float) (N - i);
		//printf("%d : %f\n", i, data[i]);
	}


	dim3 Blkdim = dim3(32 * BLK(N, 32), 1, 1);
	dim3 Grddim = dim3(1, 1, 1);

	size_t sm_size =  BLK(N, 32) * sizeof(float);
	reduction_min_v2 <<< Grddim, Blkdim, sm_size >>> (data, BLK(N, 32), result);

	print_2d_array(result, 1, 1, "result");

	// release
	hipHostFree(data);
	hipHostFree(result);
}


int main(int argc, char *argv[])
{
	int cuda_device = 0;

	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
	cuda_device = findCudaDevice(argc, (const char **)argv);

	hipDeviceProp_t deviceProp;
	checkCudaErrors(hipGetDevice(&cuda_device));
	checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));
	printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n\n",
			deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);


	// __shfl intrinsic needs SM 3.0 or higher
	if (deviceProp.major < 3)
	{
		printf("> __shfl() intrinsic requires device SM 3.0+\n");
		printf("> Waiving test.\n");
		exit(EXIT_SUCCESS);
	}


	gpu_min_test1();


	gpu_min_test2();



	hipDeviceReset();
}
