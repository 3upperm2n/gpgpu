#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


inline
hipError_t checkCuda(hipError_t result)
{
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %sn", 
				hipGetErrorString(result));
		assert(result == hipSuccess);
	}
	return result;
}





int main(int argc, char **argv) {

	int devid;

	if(argc > 2) {
		fprintf(stderr,"too many args! specify just the device to query.\n");
		exit(1);
	} else if(argc == 2) {
		devid = atoi(argv[1]);
	} else {
		devid = 0;	
	}

	hipDeviceProp_t prop;
	checkCuda( hipGetDeviceProperties(&prop, devid) );
	printf("Device: %s\n", prop.name);

	printf("Local L1 Cache Supported  : %d\n", prop.localL1CacheSupported);
	printf("Global L1 Cache Supported : %d\n", prop.globalL1CacheSupported);

	hipDeviceReset();

	return 0;
}
