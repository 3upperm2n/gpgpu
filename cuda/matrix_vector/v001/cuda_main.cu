#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>     /* strtok() */
#include <sys/types.h>  /* open() */
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>     /* getopt() */

#include <iostream>

#include <hip/hip_runtime.h>                                                       
#include <hip/hip_runtime_api.h> 
#include <helper_functions.h>   

#define FLT_SIZE sizeof(float)

using namespace std;

void test_v1a(int rows, int cols);

void init2D(float *array, int rows, int cols, float value)
{                                                                               
	for(int i=0; i<rows; i++) {                                                 
		for(int j=0; j<cols; j++) {                                             
			array[i * cols + j] = value;                                        
		}                                                                       
	}                                                                           
}

void print2D(float *array, int rows, int cols)
{
	printf("\n");
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			printf("%5.3f ", array[i * cols + j]);
		}
		printf("\n");
	}                                                                           
}

void init1D(float *data, int len, float value)
{                                                                               
	for(int i=0; i<len; i++) {                                                 
		data[i] = value;                                        
	}                                                                           
}

void print1D(float *data, int len)
{                                                                               
	printf("\n");
	for(int i=0; i<len; i++) {                                                 
		printf("%5.3f ", data[i]);
	}                                                                           
	printf("\n");
}

void d2h_print1d(float *d_data, float *h_data, const int rows)
{
	hipMemcpy(h_data, d_data, sizeof(float) * rows, hipMemcpyDeviceToHost);
	for(int i=0; i<rows; i++) {
		printf("%f ", h_data[i]);
	}
	printf("\n");
}

int check(float *d_data, float *h_data, const int rows, const int cols)
{
	float cpu = cols * 0.02;
	hipMemcpy(h_data, d_data, sizeof(float) * rows, hipMemcpyDeviceToHost);

	int correct = 1;
	for(int i=0; i<rows; i++) {
		if(h_data[i] != cpu) {
			fprintf(stderr, "result doesn't match! pos : %d, gpu %f , cpu %f\n", 
					i, h_data[i], cpu);
			correct = 0;
			break;
		}
	}
	return correct;
}

void h2d_copy(float *h_data, float *d_data, int len)
{
	hipMemcpy(d_data, h_data, sizeof(float) * len, hipMemcpyHostToDevice);
}

// timer
//double timing, runtime;
// seconds 
//extern double wtime(void);

inline int BLK(int number, int blksize)                                         
{                                                                               
    return (number + blksize - 1) / blksize;                                    
}                                                                               

// constant memory
//__constant__ float const_mem[16000];

//----------------------------------------------------------------------------//
// 
//----------------------------------------------------------------------------//
__global__ void kernel_sgemv_v1a (const int rows,
		const int cols,
		const int col_iters,
		const float* __restrict__ A,
		const float* __restrict__ B,
		float* __restrict__ C)
{
	int gx = threadIdx.x;
	int gy = threadIdx.y + __mul24(blockIdx.y, blockDim.y); // rows

	int lane_id = threadIdx.x & 0x1F;

	float tmp = 0.f;
	int row_idx = gy * cols;
	for(int i=0; i<col_iters; i++)
	{
		int curr_col = gx + i * 32;
		if (curr_col < cols)
			tmp += A[row_idx + curr_col] * B[curr_col];
	}

	// warp reduction on tmp	
	tmp += __shfl_down(tmp, 16, 32);                                      
	tmp += __shfl_down(tmp,  8, 32);                                      
	tmp += __shfl_down(tmp,  4, 32);                                      
	tmp += __shfl_down(tmp,  2, 32);                                      
	tmp += __shfl_down(tmp,  1, 32);                                      

	if(lane_id == 0) {
		C[gy] = tmp;
	}
}


void test_v1a(int rows, int cols)
{
	hipEvent_t startEvent, stopEvent;
	checkCudaErrors( hipEventCreate(&startEvent) );
	checkCudaErrors( hipEventCreate(&stopEvent) );
	// host
	float *A;
	float *B;
	float *C;
	checkCudaErrors(hipHostMalloc((void **)&A, 	rows * cols * FLT_SIZE));
	checkCudaErrors(hipHostMalloc((void **)&B, 	cols * FLT_SIZE));
	checkCudaErrors(hipHostMalloc((void **)&C, 	rows * FLT_SIZE));
	// init
	init2D(A, rows, cols, 0.2f);
	init1D(B, cols, 0.1f);
	// dump
	//print2D(A, rows, cols);
	//print1D(B, cols);
	// device
	float *d_A;
	float *d_B;
	float *d_C;
	checkCudaErrors(hipMalloc((void **)&d_A, 	rows * cols * FLT_SIZE));
	checkCudaErrors(hipMalloc((void **)&d_B, 	cols * FLT_SIZE));
	checkCudaErrors(hipMalloc((void **)&d_C, 	rows * FLT_SIZE));
	// copy data to device
	h2d_copy(A, d_A, rows * cols);
	h2d_copy(B, d_B, cols);
	h2d_copy(C, d_C, cols);
	// start gpu timing
	hipEventRecord(startEvent);
	//--------------------------------------------------------------------------
	// kernel
	//--------------------------------------------------------------------------
    dim3 Blk_config = dim3(32, 4, 1);                                           
    dim3 Grd_config = dim3(1, BLK(rows, 4), 1);

	kernel_sgemv_v1a <<< Grd_config, Blk_config>>>(rows, 
			cols, 
			BLK(cols,32),
			d_A,
			d_B,
			d_C);
	// end of gpu timing
	hipEventRecord(stopEvent);
	hipEventSynchronize(stopEvent); 

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, startEvent, stopEvent);
	cout << milliseconds << " (ms)" << endl;

	//d2h_print1d(d_C, C, rows);
	if (check(d_C, C, rows, cols))	{
		printf("success!\n");
	}


	// release
	if (A != NULL)				checkCudaErrors(hipHostFree(A));
	if (B != NULL)				checkCudaErrors(hipHostFree(B));
	if (C != NULL)				checkCudaErrors(hipHostFree(C));

	if (d_A != NULL)			checkCudaErrors(hipFree(d_A));
	if (d_B != NULL) 			checkCudaErrors(hipFree(d_B));
	if (d_C != NULL)			checkCudaErrors(hipFree(d_C));
}

int main(int argc, char **argv) {

	hipDeviceProp_t prop;
	checkCudaErrors( hipGetDeviceProperties(&prop, 0) );
	printf("Device: %s\n", prop.name);


	// 10K
	//test(100,   100);
	
	//------------------------------------------------------------------------//
	// case study 1
	//------------------------------------------------------------------------//

	// lanch a 2d grid, where x is on column with fixed warp size 32
	//test_v1a(50,   50);

	// warm-up
	test_v1a(100,   50);
	test_v1a(100,   50);
	//test_v1a(1000,   50);
	//test_v1a(100,   100);

    return(0);
}

