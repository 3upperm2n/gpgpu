#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>     /* strtok() */
#include <sys/types.h>  /* open() */
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>     /* getopt() */

#include <iostream>

#include <hip/hip_runtime.h>                                                       
#include <hip/hip_runtime_api.h> 
#include <helper_functions.h>   

#define FLT_SIZE sizeof(float)

using namespace std;

void test_v1a(int rows, int cols);

void init2D(float *array, int rows, int cols, float value)
{                                                                               
	for(int i=0; i<rows; i++) {                                                 
		for(int j=0; j<cols; j++) {                                             
			array[i * cols + j] = value;                                        
		}                                                                       
	}                                                                           
}

void print2D(float *array, int rows, int cols)
{
	printf("\n");
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			printf("%5.3f ", array[i * cols + j]);
		}
		printf("\n");
	}                                                                           
}

void init1D(float *data, int len, float value)
{                                                                               
	for(int i=0; i<len; i++) {                                                 
		data[i] = value;                                        
	}                                                                           
}

void print1D(float *data, int len)
{                                                                               
	printf("\n");
	for(int i=0; i<len; i++) {                                                 
		printf("%5.3f ", data[i]);
	}                                                                           
	printf("\n");
}

void d2h_print1d(float *d_data, float *h_data, const int rows)
{
	hipMemcpy(h_data, d_data, sizeof(float) * rows, hipMemcpyDeviceToHost);
	for(int i=0; i<rows; i++) {
		printf("%f ", h_data[i]);
	}
	printf("\n");
}

int check(float *d_data, float *h_data, const int rows, const int cols)
{
	float cpu = cols * 0.02;
	hipMemcpy(h_data, d_data, sizeof(float) * rows, hipMemcpyDeviceToHost);

	int correct = 1;
	for(int i=0; i<rows; i++) {
		//if(h_data[i] != cpu) {
		if(fabs(h_data[i] - cpu) > 1e-5) {
			fprintf(stderr, "result doesn't match! pos : %d, gpu %12.8f , cpu %12.8f\n", 
					i, h_data[i], cpu);
			correct = 0;
			break;
		}
	}
	return correct;
}

void h2d_copy(float *h_data, float *d_data, const int len)
{
	hipMemcpy(d_data, h_data, sizeof(float) * len, hipMemcpyHostToDevice);
}

// timer
//double timing, runtime;
// seconds 
//extern double wtime(void);

inline int BLK(int number, int blksize)                                         
{                                                                               
    return (number + blksize - 1) / blksize;                                    
}                                                                               

//----------------------------------------------------------------------------//
// description
// 	each block work on 4 rows
//----------------------------------------------------------------------------//
__global__ void kernel_sgemv_1b128bs (const int rows,
		const int cols,
		const int col_iters,
		const int row_iters,
		const float* __restrict__ A,
		const float* __restrict__ B,
		float* __restrict__ C)
{
	__shared__ float B_sm[128];
	__shared__ float out_sm[128];

	// 128 block thread = 4 (rows) x 32 (cols)
	//int gx = threadIdx.x + __mul24(blockIdx.x, blockDim.x);
	int lx = threadIdx.x;
	int lane_id = threadIdx.x & 0x1F;
	int warp_id = threadIdx.x >> 5;			// each warp  = one row

	//printf("col_iters = %d \n", col_iters);
	int loc_c  = lane_id; 
	int loc_c1 = lane_id + 32;
	int loc_c2 = lane_id + 64;
	int loc_c3 = lane_id + 96;


	for(int i=0; i<row_iters; i++)
	{
		//int row_offset = i * 4; 
		int row_offset = (i<<2); 
		// row_offset + warp i
		int row_id = row_offset + warp_id;
		int row_idx = row_id * cols;
	
		//--------------------------------------//
		// each column iteration  = 128 cols
		//--------------------------------------//
		float tmp = 0.f;
		for (int j=0; j<col_iters; j++)
		{
			//int offset = j * 128;
			int col_offset = (j<<7);
			// all the 128 threads of current block load 128 data points from B to B_sm
			int col_iter = lx + col_offset; 
			if(col_iter < cols)
				B_sm[lx] = B[col_iter];
			__syncthreads();

			int col_idx  = loc_c  + col_offset;
			int col_idx1 = loc_c1 + col_offset; 
			int col_idx2 = loc_c2 + col_offset; 
			int col_idx3 = loc_c3 + col_offset; 

			if(row_id < rows)
			{
				// work on 1st 32 threads/cols
				// work on 2nd 32 threads/cols
				// work on 3rd 32 threads/cols
				// work on 4th 32 threads/cols
				if(col_idx  < cols) tmp += A[row_idx + col_idx]  * B_sm[loc_c];
				if(col_idx1 < cols) tmp += A[row_idx + col_idx1] * B_sm[loc_c1];
				if(col_idx2 < cols) tmp += A[row_idx + col_idx2] * B_sm[loc_c2];
				if(col_idx3 < cols) tmp += A[row_idx + col_idx3] * B_sm[loc_c3];
			}

		} // end column iteration

		// warp reduction
		tmp  += __shfl_down(tmp,  16, 32);                                      
		tmp  += __shfl_down(tmp,   8, 32);                                      
		tmp  += __shfl_down(tmp,   4, 32);                                      
		tmp  += __shfl_down(tmp,   2, 32);                                      
		tmp  += __shfl_down(tmp,   1, 32);                                      

		// each warp output 1 row data 
		if(lane_id == 0) {
			out_sm[row_id] = tmp;	
		}
	} // end row iterations

	// output
	if(lx < rows) {
		C[lx] = out_sm[lx];
	}
}


void test_v1a(int rows, int cols)
{
	hipDeviceProp_t prop;
	checkCudaErrors( hipGetDeviceProperties(&prop, 0) );
	//int sm_num =  prop.multiProcessorCount;
	//printf("sm : %d\n", sm_num);

	hipEvent_t startEvent, stopEvent;
	checkCudaErrors( hipEventCreate(&startEvent) );
	checkCudaErrors( hipEventCreate(&stopEvent) );

	// host
	float *A;
	float *B;
	float *C;
	checkCudaErrors(hipHostMalloc((void **)&A, 	rows * cols * FLT_SIZE));
	checkCudaErrors(hipHostMalloc((void **)&B, 	cols * FLT_SIZE));
	checkCudaErrors(hipHostMalloc((void **)&C, 	rows * FLT_SIZE));
	// init
	init2D(A, rows, cols, 0.2f);
	init1D(B, cols, 0.1f);
	// dump
	//print2D(A, rows, cols);
	//print1D(B, cols);
	// device
	float *d_A;
	float *d_B;
	float *d_C;
	checkCudaErrors(hipMalloc((void **)&d_A, 	rows * cols * FLT_SIZE));
	checkCudaErrors(hipMalloc((void **)&d_B, 	cols * FLT_SIZE));
	checkCudaErrors(hipMalloc((void **)&d_C, 	rows * FLT_SIZE));
	// copy data to device
	h2d_copy(A, d_A, rows * cols);
	h2d_copy(B, d_B, cols);
	//h2d_copy(C, d_C, rows);

	// start gpu timing
	hipEventRecord(startEvent);
	//--------------------------------------------------------------------------
	// kernel
	// 	each block for one row of A
	//--------------------------------------------------------------------------

	// bs 128, max 16 blks per sm
	// gtx 970 has 13 smx
	// current launch 1 block : 128 threads
    dim3 Blk_config = dim3(128, 1, 1);                                           

	// compute how many rows to launch
	//int batch_work = BLK(rows,4);
    //dim3 Grd_config = dim3(batch_work, 1, 1);
    dim3 Grd_config = dim3(1, 1, 1);


	//printf("iters: %d\n", BLK(cols, 4));

	kernel_sgemv_1b128bs <<< Grd_config, Blk_config>>>(rows, 
			cols, 
			BLK(cols, 128), // col_iter
			BLK(rows, 4),   // row_iter
			d_A,
			d_B,
			d_C);

	// end of gpu timing
	hipEventRecord(stopEvent);
	hipEventSynchronize(stopEvent); 

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, startEvent, stopEvent);
	cout << milliseconds << " (ms)" << endl;

	//d2h_print1d(d_C, C, rows);
	if (check(d_C, C, rows, cols))	{
		printf("success!\n");
	}


	// release
	if (A != NULL)				checkCudaErrors(hipHostFree(A));
	if (B != NULL)				checkCudaErrors(hipHostFree(B));
	if (C != NULL)				checkCudaErrors(hipHostFree(C));

	if (d_A != NULL)			checkCudaErrors(hipFree(d_A));
	if (d_B != NULL) 			checkCudaErrors(hipFree(d_B));
	if (d_C != NULL)			checkCudaErrors(hipFree(d_C));

	hipDeviceReset();
}



int main(int argc, char **argv) {

	hipDeviceProp_t prop;
	checkCudaErrors( hipGetDeviceProperties(&prop, 0) );
	printf("Device: %s\n", prop.name);

	int rows = atoi(argv[1]);
	int cols = atoi(argv[2]);

	// 32 x 100
//	for(int i=0; i<10; i++)
//		test_v1a(rows,   cols);
//
	test_v1a(rows,   cols);

    return(0);
}
