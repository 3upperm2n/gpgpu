#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>     /* strtok() */
#include <sys/types.h>  /* open() */
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>     /* getopt() */

#include <iostream>

#include <hip/hip_runtime.h>                                                       
#include <hip/hip_runtime_api.h> 
#include <helper_functions.h>   

#define FLT_SIZE sizeof(float)

using namespace std;

void test_v1a(int rows, int cols);

void init2D(float *array, int rows, int cols, float value)
{                                                                               
	for(int i=0; i<rows; i++) {                                                 
		for(int j=0; j<cols; j++) {                                             
			array[i * cols + j] = value;                                        
		}                                                                       
	}                                                                           
}

void print2D(float *array, int rows, int cols)
{
	printf("\n");
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			printf("%5.3f ", array[i * cols + j]);
		}
		printf("\n");
	}                                                                           
}

void init1D(float *data, int len, float value)
{                                                                               
	for(int i=0; i<len; i++) {                                                 
		data[i] = value;                                        
	}                                                                           
}

void print1D(float *data, int len)
{                                                                               
	printf("\n");
	for(int i=0; i<len; i++) {                                                 
		printf("%5.3f ", data[i]);
	}                                                                           
	printf("\n");
}

void d2h_print1d(float *d_data, float *h_data, const int rows)
{
	hipMemcpy(h_data, d_data, sizeof(float) * rows, hipMemcpyDeviceToHost);
	for(int i=0; i<rows; i++) {
		printf("%f ", h_data[i]);
	}
	printf("\n");
}

int check(float *d_data, float *h_data, const int rows, const int cols)
{
	float cpu = cols * 0.02;
	hipMemcpy(h_data, d_data, sizeof(float) * rows, hipMemcpyDeviceToHost);

	int correct = 1;
	for(int i=0; i<rows; i++) {
		//if(h_data[i] != cpu) {
		if(fabs(h_data[i] - cpu) > 1e-5) {
			fprintf(stderr, "result doesn't match! pos : %d, gpu %12.8f , cpu %12.8f\n", 
					i, h_data[i], cpu);
			correct = 0;
			break;
		}
	}
	return correct;
}

void h2d_copy(float *h_data, float *d_data, const int len)
{
	hipMemcpy(d_data, h_data, sizeof(float) * len, hipMemcpyHostToDevice);
}

// timer
//double timing, runtime;
// seconds 
//extern double wtime(void);

inline int BLK(int number, int blksize)                                         
{                                                                               
    return (number + blksize - 1) / blksize;                                    
}                                                                               

// constant memory
//__constant__ float const_mem[16000];

//----------------------------------------------------------------------------//
// tile A: 2
//----------------------------------------------------------------------------//
__global__ void kernel_sgemv_1d1024b (const int rows,
		const int cols,
		const float* __restrict__ A,
		const float* __restrict__ B,
		float* __restrict__ C)
{
	// 128 = 4 warps
	// 256 = 8 warps
	// 512 = 16 warps
	// 1024 = 32 warps
	__shared__ float sb[32];

	int gx = threadIdx.x + __mul24(blockIdx.x, blockDim.x);
	int lx = threadIdx.x;
	int bx = blockIdx.x;	// 1 block for 1 row

	// lx % 32
	int lane_id = threadIdx.x & 0x1F;

	// lx / 32
	int warp_id = threadIdx.x >> 5;
	
	float c = 0.f;

	if(lx < cols) {
		c = A[bx * cols + lx] * B[lx];
	}

	// each warp do reduction
	c += __shfl_down(c, 16, 32);                                      
	c += __shfl_down(c,  8, 32);                                      
	c += __shfl_down(c,  4, 32);                                      
	c += __shfl_down(c,  2, 32);                                      
	c += __shfl_down(c,  1, 32);  


	// 32 warps  = 32 data points
	if(lane_id == 0) {
		sb[warp_id] = c;	
	}

	__syncthreads();

	float tmp = 0.f;

	if(warp_id == 0) {
		tmp = sb[lx];

		tmp += __shfl_down(tmp,  16,32);                                      
		tmp += __shfl_down(tmp,  8, 32);                                      
		tmp += __shfl_down(tmp,  4, 32);                                      
		tmp += __shfl_down(tmp,  2, 32);                                      
		tmp += __shfl_down(tmp,  1, 32);  
	}

	if(lx == 0) {
		C[bx] = tmp; 
	}
}


void test_v1a(int rows, int cols)
{
	hipEvent_t startEvent, stopEvent;
	checkCudaErrors( hipEventCreate(&startEvent) );
	checkCudaErrors( hipEventCreate(&stopEvent) );

	// host
	float *A;
	float *B;
	float *C;
	checkCudaErrors(hipHostMalloc((void **)&A, 	rows * cols * FLT_SIZE));
	checkCudaErrors(hipHostMalloc((void **)&B, 	cols * FLT_SIZE));
	checkCudaErrors(hipHostMalloc((void **)&C, 	rows * FLT_SIZE));
	// init
	init2D(A, rows, cols, 0.2f);
	init1D(B, cols, 0.1f);
	// dump
	//print2D(A, rows, cols);
	//print1D(B, cols);
	// device
	float *d_A;
	float *d_B;
	float *d_C;
	checkCudaErrors(hipMalloc((void **)&d_A, 	rows * cols * FLT_SIZE));
	checkCudaErrors(hipMalloc((void **)&d_B, 	cols * FLT_SIZE));
	checkCudaErrors(hipMalloc((void **)&d_C, 	rows * FLT_SIZE));
	// copy data to device
	h2d_copy(A, d_A, rows * cols);
	h2d_copy(B, d_B, cols);
	//h2d_copy(C, d_C, rows);

	// start gpu timing
	hipEventRecord(startEvent);
	//--------------------------------------------------------------------------
	// kernel
	// 	each block for one row of A
	//--------------------------------------------------------------------------
    dim3 Blk_config = dim3(1024, 1, 1);                                           
    dim3 Grd_config = dim3(rows, 1, 1);

	//printf("iters: %d\n", BLK(cols, 4));

	kernel_sgemv_1d1024b <<< Grd_config, Blk_config>>>(rows, 
			cols, 
			d_A,
			d_B,
			d_C);

	// end of gpu timing
	hipEventRecord(stopEvent);
	hipEventSynchronize(stopEvent); 

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, startEvent, stopEvent);
	cout << milliseconds << " (ms)" << endl;

	//d2h_print1d(d_C, C, rows);
	if (check(d_C, C, rows, cols))	{
		printf("success!\n");
	}


	// release
	if (A != NULL)				checkCudaErrors(hipHostFree(A));
	if (B != NULL)				checkCudaErrors(hipHostFree(B));
	if (C != NULL)				checkCudaErrors(hipHostFree(C));

	if (d_A != NULL)			checkCudaErrors(hipFree(d_A));
	if (d_B != NULL) 			checkCudaErrors(hipFree(d_B));
	if (d_C != NULL)			checkCudaErrors(hipFree(d_C));

	hipDeviceReset();
}



int main(int argc, char **argv) {

	hipDeviceProp_t prop;
	checkCudaErrors( hipGetDeviceProperties(&prop, 0) );
	printf("Device: %s\n", prop.name);

	// bs 512
	int rows = atoi(argv[1]);
	int cols = atoi(argv[2]);

	for(int i=0; i<10; i++)
		test_v1a(rows,   cols);

	test_v1a(rows,   cols);

	//test_v1a(5,   6);
	//test_v1a(100,   200);
	//test_v1a(256,   512);
	//test_v1a(1000,   1000);

	
	//------------------------------------------------------------------------//
	// case study 1
	//------------------------------------------------------------------------//

	// lanch a 2d grid, where x is on column with fixed warp size 32
	//test_v1a(50,   50);

	// warm-up
	//test_v1a(100,   50);
	//test_v1a(100,   50);
	//test_v1a(1000,   50);
	//test_v1a(100,   100);

    return(0);
}
