#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>     /* strtok() */
#include <sys/types.h>  /* open() */
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>     /* getopt() */

#include <iostream>

#include <hip/hip_runtime.h>                                                       
#include <hip/hip_runtime_api.h> 
#include <helper_functions.h>   

#define FLT_SIZE sizeof(float)

using namespace std;

void test_v1a(int rows, int cols);

void init2D(float *array, int rows, int cols, float value)
{                                                                               
	for(int i=0; i<rows; i++) {                                                 
		for(int j=0; j<cols; j++) {                                             
			array[i * cols + j] = value;                                        
		}                                                                       
	}                                                                           
}

void print2D(float *array, int rows, int cols)
{
	printf("\n");
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			printf("%5.3f ", array[i * cols + j]);
		}
		printf("\n");
	}                                                                           
}

void init1D(float *data, int len, float value)
{                                                                               
	for(int i=0; i<len; i++) {                                                 
		data[i] = value;                                        
	}                                                                           
}

void print1D(float *data, int len)
{                                                                               
	printf("\n");
	for(int i=0; i<len; i++) {                                                 
		printf("%5.3f ", data[i]);
	}                                                                           
	printf("\n");
}

void d2h_print1d(float *d_data, float *h_data, const int rows)
{
	hipMemcpy(h_data, d_data, sizeof(float) * rows, hipMemcpyDeviceToHost);
	for(int i=0; i<rows; i++) {
		printf("%f ", h_data[i]);
	}
	printf("\n");
}

int check(float *d_data, float *h_data, const int rows, const int cols)
{
	float cpu = cols * 0.02;
	hipMemcpy(h_data, d_data, sizeof(float) * rows, hipMemcpyDeviceToHost);

	int correct = 1;
	for(int i=0; i<rows; i++) {
		if(h_data[i] != cpu) {
			fprintf(stderr, "result doesn't match! pos : %d, gpu %f , cpu %f\n", 
					i, h_data[i], cpu);
			correct = 0;
			break;
		}
	}
	return correct;
}

void h2d_copy(float *h_data, float *d_data, int len)
{
	hipMemcpy(d_data, h_data, sizeof(float) * len, hipMemcpyHostToDevice);
}

// timer
//double timing, runtime;
// seconds 
//extern double wtime(void);

inline int BLK(int number, int blksize)                                         
{                                                                               
    return (number + blksize - 1) / blksize;                                    
}                                                                               

// constant memory
//__constant__ float const_mem[16000];

//----------------------------------------------------------------------------//
// 
//----------------------------------------------------------------------------//
__global__ void kernel_sgemv_v1a (const int rows,
		const int cols,
		const int col_iters,
		const float* __restrict__ A,
		const float* __restrict__ B,
		float* __restrict__ C)
{
	int gx  = threadIdx.x;
	int gy  = threadIdx.y + __mul24(blockIdx.y, blockDim.y); // rows

	// 4x work
	gy = (gy << 2);

	int lane_id = threadIdx.x & 0x1F;

	int row_idx  = gy * cols;

	float tmp = 0.f;
	float tmp1 = 0.f;
	float tmp2 = 0.f;
	float tmp3 = 0.f;

	int stride1 = cols;
	int stride2 = (cols<<1);
	int stride3 = stride2 + cols;

	//printf("col iter : %d\n", col_iters);

	// each iteration, x4 work
	for(int i=0; i<col_iters; i++)
	{
		//int curr_col  = gx + i * 128;
		int curr_col  = gx + (i<<7);
		int curr_col1 = curr_col + 32;
		int curr_col2 = curr_col + 64;
		int curr_col3 = curr_col + 96;

		float b;
		float b1;
		float b2;
		float b3;

		int addr;
		int addr1;
		int addr2;
		int addr3;

		// prefetch 1
		if (curr_col1 < cols) 
		{
			b1    = B[curr_col1];
			addr1 = row_idx + curr_col1;
		}

		// work 
		if (curr_col < cols) 
		{
			b = B[curr_col];
			addr = row_idx + curr_col;
			tmp   += A[addr]              * b;
			tmp1  += A[addr + stride1]    * b;
			tmp2  += A[addr + stride2]    * b;
			tmp3  += A[addr + stride3]    * b;
		}

		// prefetch 2
		if (curr_col2 < cols) 
		{
			b2    = B[curr_col2];
			addr2 = row_idx + curr_col2;
		}

		// work 1
		if (curr_col1 < cols) 
		{
			tmp   += A[addr1]              * b1;
			tmp1  += A[addr1 + stride1]    * b1;
			tmp2  += A[addr1 + stride2]    * b1;
			tmp3  += A[addr1 + stride3]    * b1;
		}

		// prefetch 3
		if (curr_col3 < cols) 
		{
			b3    = B[curr_col3];
			addr3 = row_idx + curr_col3;
		}

		// work 2
		if (curr_col2 < cols) 
		{
			tmp   += A[addr2]              * b2;
			tmp1  += A[addr2 + stride1]    * b2;
			tmp2  += A[addr2 + stride2]    * b2;
			tmp3  += A[addr2 + stride3]    * b2;
		}

		// work 3
		if (curr_col3 < cols) 
		{
			tmp   += A[addr3]              * b3;
			tmp1  += A[addr3 + stride1]    * b3;
			tmp2  += A[addr3 + stride2]    * b3;
			tmp3  += A[addr3 + stride3]    * b3;
		}



		//printf("tmp %f, tmp1 %f\n", tmp, tmp1);
	}

	// warp reduction on tmp	
	tmp  += __shfl_down(tmp,  16, 32);                                      
	tmp1 += __shfl_down(tmp1, 16, 32);                                      
	tmp2 += __shfl_down(tmp2, 16, 32);                                      
	tmp3 += __shfl_down(tmp3, 16, 32);                                      

	tmp  += __shfl_down(tmp,  8, 32);                                      
	tmp1 += __shfl_down(tmp1,  8, 32);                                      
	tmp2 += __shfl_down(tmp2,  8, 32);                                      
	tmp3 += __shfl_down(tmp3,  8, 32);                                      

	tmp  += __shfl_down(tmp,  4, 32);                                      
	tmp1 += __shfl_down(tmp1,  4, 32);                                      
	tmp2 += __shfl_down(tmp2,  4, 32);                                      
	tmp3 += __shfl_down(tmp3,  4, 32);                                      

	tmp  += __shfl_down(tmp,   2, 32);                                      
	tmp1 += __shfl_down(tmp1,  2, 32);                                      
	tmp2 += __shfl_down(tmp2,  2, 32);                                      
	tmp3 += __shfl_down(tmp3,  2, 32);                                      

	tmp  += __shfl_down(tmp,   1, 32);                                      
	tmp1 += __shfl_down(tmp1,  1, 32);                                      
	tmp2 += __shfl_down(tmp2,  1, 32);                                      
	tmp3 += __shfl_down(tmp3,  1, 32);    

	if(lane_id == 0) {
		C[gy]      = tmp;
		C[gy + 1]  = tmp1;
		C[gy + 2]  = tmp2;
		C[gy + 3]  = tmp3;
	}

}


template <int CHK> void test_v1a(int rows, int cols)
{
	hipEvent_t startEvent, stopEvent;
	checkCudaErrors( hipEventCreate(&startEvent) );
	checkCudaErrors( hipEventCreate(&stopEvent) );
	// host
	float *A;
	float *B;
	float *C;
	checkCudaErrors(hipHostMalloc((void **)&A, 	rows * cols * FLT_SIZE));
	checkCudaErrors(hipHostMalloc((void **)&B, 	cols * FLT_SIZE));
	checkCudaErrors(hipHostMalloc((void **)&C, 	rows * FLT_SIZE));
	// init
	init2D(A, rows, cols, 0.2f);
	init1D(B, cols, 0.1f);
	// dump
	//print2D(A, rows, cols);
	//print1D(B, cols);
	// device
	float *d_A;
	float *d_B;
	float *d_C;
	checkCudaErrors(hipMalloc((void **)&d_A, 	rows * cols * FLT_SIZE));
	checkCudaErrors(hipMalloc((void **)&d_B, 	cols * FLT_SIZE));
	checkCudaErrors(hipMalloc((void **)&d_C, 	rows * FLT_SIZE));
	// copy data to device
	h2d_copy(A, d_A, rows * cols);
	h2d_copy(B, d_B, cols);
	h2d_copy(C, d_C, cols);
	// start gpu timing
	hipEventRecord(startEvent);
	//--------------------------------------------------------------------------
	// kernel
	//--------------------------------------------------------------------------
    dim3 Blk_config = dim3(32, 4, 1);                                           
    dim3 Grd_config = dim3(1, BLK(rows/4, 4), 1);

	kernel_sgemv_v1a <<< Grd_config, Blk_config>>>(rows, 
			cols, 
			//BLK(cols,32),
			//BLK(cols,64),
			BLK(cols, 128),
			d_A,
			d_B,
			d_C);
	// end of gpu timing
	hipEventRecord(stopEvent);
	hipEventSynchronize(stopEvent); 

	if(CHK)
	{
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, startEvent, stopEvent);
		//cout << milliseconds << " (ms)" << endl;
		printf("%f (ms)\n", milliseconds);
	}

	/*
	//d2h_print1d(d_C, C, rows);
	if (check(d_C, C, rows, cols))	{
		printf("success!\n");
	}
	*/


	// release
	if (A != NULL)				checkCudaErrors(hipHostFree(A));
	if (B != NULL)				checkCudaErrors(hipHostFree(B));
	if (C != NULL)				checkCudaErrors(hipHostFree(C));

	if (d_A != NULL)			checkCudaErrors(hipFree(d_A));
	if (d_B != NULL) 			checkCudaErrors(hipFree(d_B));
	if (d_C != NULL)			checkCudaErrors(hipFree(d_C));
}

int main(int argc, char **argv) {

	//hipDeviceProp_t prop;
	//checkCudaErrors( hipGetDeviceProperties(&prop, 0) );
	//printf("Device: %s\n", prop.name);

	int rows = atoi(argv[1]);                                                   
	int cols = atoi(argv[2]);                                                   
	//printf("rows %d, cols %d\n", rows, cols);

	// 10K
	//test(100,   100);
	
	//------------------------------------------------------------------------//
	// case study 1
	//------------------------------------------------------------------------//

	// lanch a 2d grid, where x is on column with fixed warp size 32
	//test_v1a(50,   50);

	// warm-up                                                                  
	for(int i=0; i<10; i++)                                                     
		test_v1a<0>(rows,   cols);                                                  

	test_v1a<1>(rows,   cols); 



	///test_v1a(100,   50);
	//test_v1a(1000,   50);
	//test_v1a(100,   100);

    return(0);
}

