#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>     /* strtok() */
#include <sys/types.h>  /* open() */
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>     /* getopt() */

#include <math.h>

#include <iostream>

#include <hip/hip_runtime.h>                                                       
#include <hip/hip_runtime_api.h> 
#include <helper_functions.h>   
#include <hip/hip_vector_types.h>   

#define FLT_SIZE sizeof(float)

using namespace std;

void test_v1a(int rows, int cols);

void init2D(float *array, int rows, int cols, float value)
{                                                                               
	for(int i=0; i<rows; i++) {                                                 
		for(int j=0; j<cols; j++) {                                             
			array[i * cols + j] = value;                                        
		}                                                                       
	}                                                                           
}

void print2D(float *array, int rows, int cols)
{
	printf("\n");
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			printf("%5.3f ", array[i * cols + j]);
		}
		printf("\n");
	}                                                                           
}

void init1D(float *data, int len, float value)
{                                                                               
	for(int i=0; i<len; i++) {                                                 
		data[i] = value;                                        
	}                                                                           
}

void print1D(float *data, int len)
{                                                                               
	printf("\n");
	for(int i=0; i<len; i++) {                                                 
		printf("%5.3f ", data[i]);
	}                                                                           
	printf("\n");
}

void d2h_print1d(float *d_data, float *h_data, const int rows)
{
	hipMemcpy(h_data, d_data, sizeof(float) * rows, hipMemcpyDeviceToHost);
	for(int i=0; i<rows; i++) {
		printf("%f ", h_data[i]);
	}
	printf("\n");
}

int check(float *d_data, float *h_data, const int rows, const int cols)
{
	float cpu = cols * 0.02;
	hipMemcpy(h_data, d_data, sizeof(float) * rows, hipMemcpyDeviceToHost);

	int correct = 1;
	for(int i=0; i<rows; i++) {
		//if(h_data[i] != cpu) {
		if(fabs(h_data[i] -cpu) > 1e-4) {
			fprintf(stderr, "result doesn't match! pos : %d, gpu %f , cpu %f\n", 
					i, h_data[i], cpu);
			correct = 0;
			break;
		}
	}
	return correct;
}

void h2d_copy(float *h_data, float *d_data, int len)
{
	hipMemcpy(d_data, h_data, sizeof(float) * len, hipMemcpyHostToDevice);
}

// timer
//double timing, runtime;
// seconds 
//extern double wtime(void);

inline int BLK(int number, int blksize)                                         
{                                                                               
    return (number + blksize - 1) / blksize;                                    
}                                                                               

// constant memory
//__constant__ float const_mem[16000];

//----------------------------------------------------------------------------//
// 
//----------------------------------------------------------------------------//
__global__ void kernel_sgemv_v1a (const int rows,
		const int cols,
		const int col_iters,
		const float* __restrict__ A,
		const float* __restrict__ B,
		float* __restrict__ C)
{
	int gx  = threadIdx.x;
	int gy  = threadIdx.y + __mul24(blockIdx.y, blockDim.y); // rows

	// 2x work
	gy = (gy << 1);

	int lane_id = threadIdx.x & 0x1F;

	int row_idx  = gy * cols;

	//float tmp = 0.f;
	//float tmp1 = 0.f;

	float2 tmp = make_float2(0.f, 0.f);

	float2 A_vec;
	float2 preA;
	float2 preA1;

	//printf("col iter : %d\n", col_iters);

	// each iteration, x4 work
	for(int i=0; i<col_iters; i++)
	{
		//int curr_col  = gx + i * 128;
		int curr_col  = gx + (i<<7);
		int curr_col1 = curr_col + 32;
		int curr_col2 = curr_col + 64;
		int curr_col3 = curr_col + 96;

		float b;
		float b1;
		float b2;
		float b3;

		int addr;
		int addr1;
		int addr2;
		int addr3;


		//float preA;
		//float preA1;

		//float preA2;
		//float preA3;


		// prefetch 1
		if (curr_col1 < cols) 
		{
			b1 = B[curr_col1];
			addr1 = row_idx + curr_col1;

			//preA  = A[addr1];
			//preA1 = A[addr1 + cols];

			preA = make_float2(A[addr1], A[addr1 + cols]);
		}

		// work 
		if (curr_col < cols) 
		{
			b = B[curr_col];
			//printf("b : %f\n", b);
			addr = row_idx + curr_col;

			A_vec = make_float2(A[addr], A[addr + cols]);
			tmp += A_vec * b;

			//tmp   += A[addr]           * b;
			//tmp1  += A[addr + cols]    * b;
		}

		// prefetch 2
		if (curr_col2 < cols) 
		{
			b2    = B[curr_col2];
			addr2 = row_idx + curr_col2;

			//preA2  = A[addr2];
			//preA3  = A[addr2 + cols];
			preA1 = make_float2(A[addr2], A[addr2 + cols]);
		}

		// work 1
		if (curr_col1 < cols) 
		{
			//tmp   += A[addr1]           * b1;
			//tmp1  += A[addr1 + cols]    * b1;

			//tmp   += preA  * b1;
			//tmp1  += preA1 * b1;
			tmp += preA * b1;
		}

		// prefetch 3
		if (curr_col3 < cols) 
		{
			b3    = B[curr_col3];
			addr3 = row_idx + curr_col3;

			//preA   = A[addr3];
			//preA1  = A[addr3 + cols];
			preA = make_float2(A[addr3], A[addr3 + cols]);
		}

		// work 2
		if (curr_col2 < cols) 
		{
			//tmp   += A[addr2]           * b2;
			//tmp1  += A[addr2 + cols]    * b2;
			//tmp   += preA2 * b2;
			//tmp1  += preA3 * b2;
			tmp += preA1 * b2;
		}

		// work 3
		if (curr_col3 < cols) 
		{
			//tmp   += A[addr3]           * b3;
			//tmp1  += A[addr3 + cols]    * b3;
			//tmp   += preA  * b3;
			//tmp1  += preA1 * b3;
			tmp += preA * b3;
		}
	}

	// warp reduction on tmp	

	tmp.x  += __shfl_down(tmp.x,  16, 32);                                      
	tmp.y  += __shfl_down(tmp.y,  16, 32);                                      

	tmp.x  += __shfl_down(tmp.x,   8, 32);                                      
	tmp.y  += __shfl_down(tmp.y,   8, 32);                                      

	tmp.x  += __shfl_down(tmp.x,   4, 32);                                      
	tmp.y  += __shfl_down(tmp.y,   4, 32);                                      

	tmp.x  += __shfl_down(tmp.x,   2, 32);                                      
	tmp.y  += __shfl_down(tmp.y,   2, 32);                                      

	tmp.x  += __shfl_down(tmp.x,   1, 32);
	tmp.y  += __shfl_down(tmp.y,   1, 32);                                      

	if(lane_id == 0) {
		C[gy]      = tmp.x;
		C[gy + 1]  = tmp.y;
	}

}


template <int CHK> void test_v1a(int rows, int cols)
{
	hipEvent_t startEvent, stopEvent;
	checkCudaErrors( hipEventCreate(&startEvent) );
	checkCudaErrors( hipEventCreate(&stopEvent) );
	// host
	float *A;
	float *B;
	float *C;
	checkCudaErrors(hipHostMalloc((void **)&A, 	rows * cols * FLT_SIZE));
	checkCudaErrors(hipHostMalloc((void **)&B, 	cols * FLT_SIZE));
	checkCudaErrors(hipHostMalloc((void **)&C, 	rows * FLT_SIZE));
	// init
	init2D(A, rows, cols, 0.2f);
	init1D(B, cols, 0.1f);
	// dump
	//print2D(A, rows, cols);
	//print1D(B, cols);
	// device
	float *d_A;
	float *d_B;
	float *d_C;
	checkCudaErrors(hipMalloc((void **)&d_A, 	rows * cols * FLT_SIZE));
	checkCudaErrors(hipMalloc((void **)&d_B, 	cols * FLT_SIZE));
	checkCudaErrors(hipMalloc((void **)&d_C, 	rows * FLT_SIZE));
	// copy data to device
	h2d_copy(A, d_A, rows * cols);
	h2d_copy(B, d_B, cols);
	h2d_copy(C, d_C, cols);
	// start gpu timing
	hipEventRecord(startEvent);
	//--------------------------------------------------------------------------
	// kernel
	//--------------------------------------------------------------------------
    dim3 Blk_config = dim3(32, 4, 1);                                           
    dim3 Grd_config = dim3(1, BLK((rows+1)/2, 4), 1);

	kernel_sgemv_v1a <<< Grd_config, Blk_config>>>(rows, 
			cols, 
			//BLK(cols,32),
			//BLK(cols,64),
			BLK(cols, 128),
			d_A,
			d_B,
			d_C);
	// end of gpu timing
	hipEventRecord(stopEvent);
	hipEventSynchronize(stopEvent); 

	if(CHK)
	{
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, startEvent, stopEvent);
		//cout << milliseconds << " (ms)" << endl;
		printf("%f (ms)\n", milliseconds);
	}

	////d2h_print1d(d_C, C, rows);
	//if (check(d_C, C, rows, cols))	{
	//	printf("success!\n");
	//}


	// release
	if (A != NULL)				checkCudaErrors(hipHostFree(A));
	if (B != NULL)				checkCudaErrors(hipHostFree(B));
	if (C != NULL)				checkCudaErrors(hipHostFree(C));

	if (d_A != NULL)			checkCudaErrors(hipFree(d_A));
	if (d_B != NULL) 			checkCudaErrors(hipFree(d_B));
	if (d_C != NULL)			checkCudaErrors(hipFree(d_C));
}

int main(int argc, char **argv) {

	//hipDeviceProp_t prop;
	//checkCudaErrors( hipGetDeviceProperties(&prop, 0) );
	//printf("Device: %s\n", prop.name);

	int rows = atoi(argv[1]);                                                   
	int cols = atoi(argv[2]);                                                   
	//printf("rows %d, cols %d\n", rows, cols);

	// 10K
	//test(100,   100);
	
	//------------------------------------------------------------------------//
	// case study 1
	//------------------------------------------------------------------------//

	// lanch a 2d grid, where x is on column with fixed warp size 32
	//test_v1a(50,   50);

	// warm-up                                                                  
	for(int i=0; i<10; i++)                                                     
		test_v1a<0>(rows,   cols);                                                  

	test_v1a<1>(rows,   cols); 



	///test_v1a(100,   50);
	//test_v1a(1000,   50);
	//test_v1a(100,   100);

    return(0);
}

