#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>     /* strtok() */
#include <sys/types.h>  /* open() */
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>     /* getopt() */

#include <iostream>

#include <hip/hip_runtime.h>                                                       
#include <hip/hip_runtime_api.h> 
#include <helper_functions.h>   

#define FLT_SIZE sizeof(float)

using namespace std;

void test_v1a(int rows, int cols);

void init2D(float *array, int rows, int cols, float value)
{                                                                               
	for(int i=0; i<rows; i++) {                                                 
		for(int j=0; j<cols; j++) {                                             
			array[i * cols + j] = value;                                        
		}                                                                       
	}                                                                           
}

void print2D(float *array, int rows, int cols)
{
	printf("\n");
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			printf("%5.3f ", array[i * cols + j]);
		}
		printf("\n");
	}                                                                           
}

void init1D(float *data, int len, float value)
{                                                                               
	for(int i=0; i<len; i++) {                                                 
		data[i] = value;                                        
	}                                                                           
}

void print1D(float *data, int len)
{                                                                               
	printf("\n");
	for(int i=0; i<len; i++) {                                                 
		printf("%5.3f ", data[i]);
	}                                                                           
	printf("\n");
}

void d2h_print1d(float *d_data, float *h_data, const int rows)
{
	hipMemcpy(h_data, d_data, sizeof(float) * rows, hipMemcpyDeviceToHost);
	for(int i=0; i<rows; i++) {
		printf("%f ", h_data[i]);
	}
	printf("\n");
}

int check(float *d_data, float *h_data, const int rows, const int cols)
{
	float cpu = cols * 0.02;
	hipMemcpy(h_data, d_data, sizeof(float) * rows, hipMemcpyDeviceToHost);

	int correct = 1;
	for(int i=0; i<rows; i++) {
		//if(h_data[i] != cpu) {
		if(fabs(h_data[i] - cpu) > 1e-5) {
			fprintf(stderr, "result doesn't match! pos : %d, gpu %12.8f , cpu %12.8f\n", 
					i, h_data[i], cpu);
			correct = 0;
			break;
		}
	}
	return correct;
}

void h2d_copy(float *h_data, float *d_data, const int len)
{
	hipMemcpy(d_data, h_data, sizeof(float) * len, hipMemcpyHostToDevice);
}

// timer
//double timing, runtime;
// seconds 
//extern double wtime(void);

inline int BLK(int number, int blksize)                                         
{                                                                               
    return (number + blksize - 1) / blksize;                                    
}                                                                               

// constant memory
//__constant__ float const_mem[16000];

//----------------------------------------------------------------------------//
//  2x work per block
// 	add row_iters 
//----------------------------------------------------------------------------//
__global__ void kernel_sgemv_1d128b (const int rows,
		const int cols,
		const int col_iters,
		const int offset,
		const float* __restrict__ A,
		const float* __restrict__ B,
		float* __restrict__ C)
{
	// 128 = 4 warps
	__shared__ float sb[4];
	__shared__ float sb1[4];
	__shared__ float sb2[4];
	__shared__ float sb3[4];

	__shared__ float sb4[4];
	__shared__ float sb5[4];
	__shared__ float sb6[4];
	__shared__ float sb7[4];

	__shared__ float B_sm[128];

	// resue previous 4 A sm
	__shared__ float A_sm[128];
	__shared__ float A1_sm[128];
	__shared__ float A2_sm[128];
	__shared__ float A3_sm[128];


	int gx = threadIdx.x + __mul24(blockIdx.x, blockDim.x);
	int lx = threadIdx.x;

	int bx = blockIdx.x << 3;	// 1 block for 8 row
	int bx1 = bx + 1;
	int bx2 = bx + 2;
	int bx3 = bx + 3;

	int bx4 = bx + 4;
	int bx5 = bx + 5;
	int bx6 = bx + 6;
	int bx7 = bx + 7;

	int lane_id = threadIdx.x & 0x1F;
	int warp_id = threadIdx.x >> 5;
	
	// assume B len is max 128

	// load data to shared memory
	int startpos  = bx * cols; 
	int startpos1 = startpos + cols;  
	int startpos2 = startpos1 + cols;  
	int startpos3 = startpos2 + cols;  

	int startpos4 = startpos3 + cols;  
	int startpos5 = startpos4 + cols;  
	int startpos6 = startpos5 + cols;  
	int startpos7 = startpos6 + cols;  


	float c  = 0.f;
	float c1 = 0.f;
	float c2 = 0.f;
	float c3 = 0.f;

	float c4 = 0.f;
	float c5 = 0.f;
	float c6 = 0.f;
	float c7 = 0.f;

	//-----------------------------------------//
	// iter 1 
	//-----------------------------------------//
	B_sm[lx]  = B[lx];
	A_sm[lx]  = A[startpos  + lx];
	A1_sm[lx] = A[startpos1 + lx];
	A2_sm[lx] = A[startpos2 + lx];
	A3_sm[lx] = A[startpos3 + lx];

	__syncthreads();

	if(lx < cols) {
		if(bx3 < rows) {
			c  = A_sm[lx]  * B_sm[lx]; 
			c1  = A1_sm[lx]  * B_sm[lx]; 
			c2  = A2_sm[lx]  * B_sm[lx]; 
			c3  = A3_sm[lx]  * B_sm[lx]; 
		}
		else if (bx2 < rows) {
			c  = A_sm[lx]  * B_sm[lx]; 
			c1  = A1_sm[lx]  * B_sm[lx]; 
			c2  = A2_sm[lx]  * B_sm[lx]; 
		}
		else if (bx1 < rows) {
			c  = A_sm[lx]  * B_sm[lx]; 
			c1  = A1_sm[lx]  * B_sm[lx]; 
		}
		else if (bx < rows) {
			c  = A_sm[lx]  * B_sm[lx]; 
		}
	}
	//-----------------------------------------//
	// iter 2 
	//-----------------------------------------//
	A_sm[lx]  = A[startpos4 + lx];
	A1_sm[lx] = A[startpos5 + lx];
	A2_sm[lx] = A[startpos6 + lx];
	A3_sm[lx] = A[startpos7 + lx];
	__syncthreads();
	if(lx < cols) {
		if(bx7 < rows) {
			c4  = A_sm[lx]  * B_sm[lx]; 
			c5  = A1_sm[lx]  * B_sm[lx]; 
			c6  = A2_sm[lx]  * B_sm[lx]; 
			c7  = A3_sm[lx]  * B_sm[lx]; 
		}
		else if (bx6 < rows) {
			c4  = A_sm[lx]  * B_sm[lx]; 
			c5  = A1_sm[lx]  * B_sm[lx]; 
			c6  = A2_sm[lx]  * B_sm[lx]; 
		}
		else if (bx5 < rows) {
			c4  = A_sm[lx]  * B_sm[lx]; 
			c5  = A1_sm[lx]  * B_sm[lx]; 
		}
		else if (bx4 < rows) {
			c4  = A_sm[lx]  * B_sm[lx]; 
		}
	}




	// 128 has 4 warps
	// each warp do reduction
	c  += __shfl_down(c, 16, 32);                                      
	c1 += __shfl_down(c1, 16, 32);                                      
	c2 += __shfl_down(c2, 16, 32);                                      
	c3 += __shfl_down(c3, 16, 32);                                      

	c4 += __shfl_down(c4, 16, 32);                                      
	c5 += __shfl_down(c5, 16, 32);                                      
	c6 += __shfl_down(c6, 16, 32);                                      
	c7 += __shfl_down(c7, 16, 32);                                      

	c += __shfl_down(c,    8, 32);                                      
	c1 += __shfl_down(c1,  8, 32);                                      
	c2 += __shfl_down(c2,  8, 32);                                      
	c3 += __shfl_down(c3,  8, 32);                                      

	c4 += __shfl_down(c4,  8, 32);                                      
	c5 += __shfl_down(c5,  8, 32);                                      
	c6 += __shfl_down(c6,  8, 32);                                      
	c7 += __shfl_down(c7,  8, 32);                                      



	c += __shfl_down(c,  4, 32);                                      
	c1 += __shfl_down(c1,  4, 32);                                      
	c2 += __shfl_down(c2,  4, 32);                                      
	c3 += __shfl_down(c3,  4, 32);                                      

	c4 += __shfl_down(c4,  4, 32);                                      
	c5 += __shfl_down(c5,  4, 32);                                      
	c6 += __shfl_down(c6,  4, 32);                                      
	c7 += __shfl_down(c7,  4, 32);                                      

	c += __shfl_down(c,  2, 32);                                      
	c1 += __shfl_down(c1,  2, 32);                                      
	c2 += __shfl_down(c2,  2, 32);                                      
	c3 += __shfl_down(c3,  2, 32);                                      

	c4 += __shfl_down(c4,  2, 32);                                      
	c5 += __shfl_down(c5,  2, 32);                                      
	c6 += __shfl_down(c6,  2, 32);                                      
	c7 += __shfl_down(c7,  2, 32);                                      

	c += __shfl_down(c,  1, 32);  
	c1 += __shfl_down(c1,  1, 32);  
	c2 += __shfl_down(c2,  1, 32);  
	c3 += __shfl_down(c3,  1, 32);  

	c4 += __shfl_down(c4,  1, 32);  
	c5 += __shfl_down(c5,  1, 32);  
	c6 += __shfl_down(c6,  1, 32);  
	c7 += __shfl_down(c7,  1, 32);  


	// 4 warps  = 4 data points
	if(lane_id == 0) {
		sb[warp_id] = c;	
		sb1[warp_id] = c1;	
		sb2[warp_id] = c2;	
		sb3[warp_id] = c3;	

		sb4[warp_id] = c4;	
		sb5[warp_id] = c5;	
		sb6[warp_id] = c6;	
		sb7[warp_id] = c7;	
	}

	__syncthreads();

	if(threadIdx.x == 0) {

		if(bx < rows) {
			C[bx]  = sb[0] + sb[1] + sb[2] + sb[3];	
		}

		if(bx1 < rows) {
			C[bx1] = sb1[0] + sb1[1] + sb1[2] + sb1[3];	
		}

		if(bx2 < rows) {
			C[bx2] = sb2[0] + sb2[1] + sb2[2] + sb2[3];	
		}

		if(bx3 < rows) {
			C[bx3] = sb3[0] + sb3[1] + sb3[2] + sb3[3];	
		}
		
		if(bx4 < rows) {
			C[bx4] = sb4[0] + sb4[1] + sb4[2] + sb4[3];	
		}
		if(bx5 < rows) {
			C[bx5] = sb5[0] + sb5[1] + sb5[2] + sb5[3];	
		}
		if(bx6 < rows) {
			C[bx6] = sb6[0] + sb6[1] + sb6[2] + sb6[3];	
		}
		if(bx7 < rows) {
			C[bx7] = sb7[0] + sb7[1] + sb7[2] + sb7[3];	
		}
	}
}


void test_v1a(int rows, int cols)
{
	hipDeviceProp_t prop;
	checkCudaErrors( hipGetDeviceProperties(&prop, 0) );
	//int sm_num =  prop.multiProcessorCount;
	//printf("sm : %d\n", sm_num);

	hipEvent_t startEvent, stopEvent;
	checkCudaErrors( hipEventCreate(&startEvent) );
	checkCudaErrors( hipEventCreate(&stopEvent) );

	// host
	float *A;
	float *B;
	float *C;
	checkCudaErrors(hipHostMalloc((void **)&A, 	rows * cols * FLT_SIZE));
	checkCudaErrors(hipHostMalloc((void **)&B, 	cols * FLT_SIZE));
	checkCudaErrors(hipHostMalloc((void **)&C, 	rows * FLT_SIZE));
	// init
	init2D(A, rows, cols, 0.2f);
	init1D(B, cols, 0.1f);
	// dump
	//print2D(A, rows, cols);
	//print1D(B, cols);
	// device
	float *d_A;
	float *d_B;
	float *d_C;
	checkCudaErrors(hipMalloc((void **)&d_A, 	rows * cols * FLT_SIZE));
	checkCudaErrors(hipMalloc((void **)&d_B, 	cols * FLT_SIZE));
	checkCudaErrors(hipMalloc((void **)&d_C, 	rows * FLT_SIZE));
	// copy data to device
	h2d_copy(A, d_A, rows * cols);
	h2d_copy(B, d_B, cols);
	//h2d_copy(C, d_C, rows);

	// start gpu timing
	hipEventRecord(startEvent);
	//--------------------------------------------------------------------------
	// kernel
	// 	each block for one row of A
	//--------------------------------------------------------------------------

	// bs 128, max 16 blks per sm
	// gtx 970 has 13 smx
    dim3 Blk_config = dim3(128, 1, 1);                                           
    //dim3 Grd_config = dim3(rows, 1, 1);

	//int max_rows_per_iter = 16 * sm_num;
    //dim3 Grd_config = dim3(max_rows_per_iter, 1, 1);	// bs 128, grd 208

	// 2x work per thread/row/block
	//int iterwork = max_rows_per_iter * 2;

	// not using persistent mode
	// double the work per block/rows 
	int batch_work = BLK(rows,8);
    dim3 Grd_config = dim3(batch_work, 1, 1);
	//int offset = batch_work * cols;

	//printf("iters: %d\n", BLK(cols, 4));

	kernel_sgemv_1d128b <<< Grd_config, Blk_config>>>(rows, 
			cols, 
			BLK(cols, 128), // col_iter
			batch_work,
			d_A,
			d_B,
			d_C);

	// end of gpu timing
	hipEventRecord(stopEvent);
	hipEventSynchronize(stopEvent); 

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, startEvent, stopEvent);
	cout << milliseconds << " (ms)" << endl;

	//d2h_print1d(d_C, C, rows);
	if (check(d_C, C, rows, cols))	{
		printf("success!\n");
	}


	// release
	if (A != NULL)				checkCudaErrors(hipHostFree(A));
	if (B != NULL)				checkCudaErrors(hipHostFree(B));
	if (C != NULL)				checkCudaErrors(hipHostFree(C));

	if (d_A != NULL)			checkCudaErrors(hipFree(d_A));
	if (d_B != NULL) 			checkCudaErrors(hipFree(d_B));
	if (d_C != NULL)			checkCudaErrors(hipFree(d_C));

	hipDeviceReset();
}



int main(int argc, char **argv) {

	hipDeviceProp_t prop;
	checkCudaErrors( hipGetDeviceProperties(&prop, 0) );
	printf("Device: %s\n", prop.name);

	int rows = atoi(argv[1]);
	int cols = atoi(argv[2]);

	// 100 x 100
	for(int i=0; i<10; i++)
		test_v1a(rows,   cols);

	test_v1a(rows,   cols);

    return(0);
}
