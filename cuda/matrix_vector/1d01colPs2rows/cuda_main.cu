#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>     /* strtok() */
#include <sys/types.h>  /* open() */
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>     /* getopt() */

#include <iostream>

#include <hip/hip_runtime.h>                                                       
#include <hip/hip_runtime_api.h> 
#include <helper_functions.h>   

#define FLT_SIZE sizeof(float)

using namespace std;

void test_v1a(int rows, int cols);

void init2D(float *array, int rows, int cols, float value)
{                                                                               
	for(int i=0; i<rows; i++) {                                                 
		for(int j=0; j<cols; j++) {                                             
			array[i * cols + j] = value;                                        
		}                                                                       
	}                                                                           
}

void print2D(float *array, int rows, int cols)
{
	printf("\n");
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			printf("%5.3f ", array[i * cols + j]);
		}
		printf("\n");
	}                                                                           
}

void init1D(float *data, int len, float value)
{                                                                               
	for(int i=0; i<len; i++) {                                                 
		data[i] = value;                                        
	}                                                                           
}

void print1D(float *data, int len)
{                                                                               
	printf("\n");
	for(int i=0; i<len; i++) {                                                 
		printf("%5.3f ", data[i]);
	}                                                                           
	printf("\n");
}

void d2h_print1d(float *d_data, float *h_data, const int rows)
{
	hipMemcpy(h_data, d_data, sizeof(float) * rows, hipMemcpyDeviceToHost);
	for(int i=0; i<rows; i++) {
		printf("%f ", h_data[i]);
	}
	printf("\n");
}

int check(float *d_data, float *h_data, const int rows, const int cols)
{
	float cpu = cols * 0.02;
	hipMemcpy(h_data, d_data, sizeof(float) * rows, hipMemcpyDeviceToHost);

	int correct = 1;
	for(int i=0; i<rows; i++) {
		//if(h_data[i] != cpu) {
		if(fabs(h_data[i] - cpu) > 1e-5) {
			fprintf(stderr, "result doesn't match! pos : %d, gpu %12.8f , cpu %12.8f\n", 
					i, h_data[i], cpu);
			correct = 0;
			break;
		}
	}
	return correct;
}

void h2d_copy(float *h_data, float *d_data, const int len)
{
	hipMemcpy(d_data, h_data, sizeof(float) * len, hipMemcpyHostToDevice);
}

// timer
//double timing, runtime;
// seconds 
//extern double wtime(void);

inline int BLK(int number, int blksize)                                         
{                                                                               
    return (number + blksize - 1) / blksize;                                    
}                                                                               

// constant memory
//__constant__ float const_mem[16000];

//----------------------------------------------------------------------------//
//  2x work per block
// 	add row_iters 
//----------------------------------------------------------------------------//
__global__ void kernel_sgemv_1d128b (const int rows,
		const int cols,
		const int col_iters,
		const int row_iters,
		const int iterwork,
		const float* __restrict__ A,
		const float* __restrict__ B,
		float* __restrict__ C)
{
	// 128 = 4 warps
	__shared__ float sb[4];
	__shared__ float sb1[4];

	int gx = threadIdx.x + __mul24(blockIdx.x, blockDim.x);
	int lx = threadIdx.x;


	//int bx = blockIdx.x;	// 1 block for 1 row
	// 2x work per block
	int bx = blockIdx.x * 2;
	int bx1 = bx + 1; 


	// lx % 32
	int lane_id = threadIdx.x & 0x1F;

	// lx / 32
	int warp_id = threadIdx.x >> 5;
	
	float c  = 0.f;
	float c1 = 0.f;

	// 1st iters
	if(lx < cols) {
		c  = A[bx * cols + lx] * B[lx];
		c1 = A[bx1 * cols + lx] * B[lx];
	}

	// the rest iters
	for(int i = 1; i<col_iters; i++) {
		lx += 128; 
		if(lx < cols) {
			c  += A[bx  * cols + lx] * B[lx];
			c1 += A[bx1 * cols + lx] * B[lx];
		}
	}



	// 128 has 4 warps
	// each warp do reduction
	c  += __shfl_down(c, 16, 32);                                      
	c1 += __shfl_down(c1, 16, 32);                                      

	c += __shfl_down(c,  8, 32);                                      
	c1 += __shfl_down(c1,  8, 32);                                      

	c += __shfl_down(c,  4, 32);                                      
	c1 += __shfl_down(c1,  4, 32);                                      

	c += __shfl_down(c,  2, 32);                                      
	c1 += __shfl_down(c1,  2, 32);                                      

	c += __shfl_down(c,  1, 32);  
	c1 += __shfl_down(c1,  1, 32);  


	// 4 warps  = 4 data points
	if(lane_id == 0) {
		sb[warp_id] = c;	
		sb1[warp_id] = c1;	
	}

	__syncthreads();

	if(threadIdx.x == 0) {
		if(bx < rows) {
			C[bx]  = sb[0] + sb[1] + sb[2] + sb[3];	
		}

		if(bx1 < rows) {
			C[bx1] = sb1[0] + sb1[1] + sb1[2] + sb1[3];	
		}
	}

	// when there is more work
	for(int j=1; j<row_iters; j++)
	{
		// on gtx 970, persistent mode can run 13 x 16 = 208 blocks 
		// since I doubled the work per block, it can support 416 blocks (aka output rows)
		// if rows>416, we need to run more iterations
		//int offset = j * 416; 	
		int offset = j * iterwork; 	

		//-------------------------------------------------------------------//
		//-------------------------------------------------------------------//
		bx = blockIdx.x * 2 + offset;
		bx1 = bx + 1; 

		c  = 0.f;
		c1 = 0.f;

		// 1st iters
		if(lx < cols) {
			c  = A[bx * cols + lx] * B[lx];
			c1 = A[bx1 * cols + lx] * B[lx];
		}

		// the rest iters
		for(int i = 1; i<col_iters; i++) {
			lx += 128; 
			if(lx < cols) {
				c  += A[bx  * cols + lx] * B[lx];
				c1 += A[bx1 * cols + lx] * B[lx];
			}
		}

		// 128 has 4 warps
		// each warp do reduction
		c  += __shfl_down(c, 16, 32);                                      
		c1 += __shfl_down(c1, 16, 32);                                      

		c += __shfl_down(c,  8, 32);                                      
		c1 += __shfl_down(c1,  8, 32);                                      

		c += __shfl_down(c,  4, 32);                                      
		c1 += __shfl_down(c1,  4, 32);                                      

		c += __shfl_down(c,  2, 32);                                      
		c1 += __shfl_down(c1,  2, 32);                                      

		c += __shfl_down(c,  1, 32);  
		c1 += __shfl_down(c1,  1, 32);  


		// 4 warps  = 4 data points
		if(lane_id == 0) {
			sb[warp_id] = c;	
			sb1[warp_id] = c1;	
		}

		__syncthreads();

		if(threadIdx.x == 0) {
			if(bx < rows) {
				C[bx]  = sb[0] + sb[1] + sb[2] + sb[3];	
			}

			if(bx1 < rows) {
				C[bx1] = sb1[0] + sb1[1] + sb1[2] + sb1[3];	
			}
		}

	}
}


void test_v1a(int rows, int cols)
{
	hipDeviceProp_t prop;
	checkCudaErrors( hipGetDeviceProperties(&prop, 0) );
	int sm_num =  prop.multiProcessorCount;
	//printf("sm : %d\n", sm_num);

	hipEvent_t startEvent, stopEvent;
	checkCudaErrors( hipEventCreate(&startEvent) );
	checkCudaErrors( hipEventCreate(&stopEvent) );

	// host
	float *A;
	float *B;
	float *C;
	checkCudaErrors(hipHostMalloc((void **)&A, 	rows * cols * FLT_SIZE));
	checkCudaErrors(hipHostMalloc((void **)&B, 	cols * FLT_SIZE));
	checkCudaErrors(hipHostMalloc((void **)&C, 	rows * FLT_SIZE));
	// init
	init2D(A, rows, cols, 0.2f);
	init1D(B, cols, 0.1f);
	// dump
	//print2D(A, rows, cols);
	//print1D(B, cols);
	// device
	float *d_A;
	float *d_B;
	float *d_C;
	checkCudaErrors(hipMalloc((void **)&d_A, 	rows * cols * FLT_SIZE));
	checkCudaErrors(hipMalloc((void **)&d_B, 	cols * FLT_SIZE));
	checkCudaErrors(hipMalloc((void **)&d_C, 	rows * FLT_SIZE));
	// copy data to device
	h2d_copy(A, d_A, rows * cols);
	h2d_copy(B, d_B, cols);
	//h2d_copy(C, d_C, rows);

	// start gpu timing
	hipEventRecord(startEvent);
	//--------------------------------------------------------------------------
	// kernel
	// 	each block for one row of A
	//--------------------------------------------------------------------------

	// bs 128, max 16 blks per sm
	// gtx 970 has 13 smx
    dim3 Blk_config = dim3(128, 1, 1);                                           
    //dim3 Grd_config = dim3(rows, 1, 1);

	int max_rows_per_iter = 16 * sm_num;
    dim3 Grd_config = dim3(max_rows_per_iter, 1, 1);	// bs 128, grd 208

	// 2x work per thread/row/block
	int iterwork = max_rows_per_iter * 2;


	//printf("iters: %d\n", BLK(cols, 4));

	kernel_sgemv_1d128b <<< Grd_config, Blk_config>>>(rows, 
			cols, 
			BLK(cols, 128), // col_iter
			BLK(rows, iterwork),	// row_iter
			iterwork,
			d_A,
			d_B,
			d_C);

	// end of gpu timing
	hipEventRecord(stopEvent);
	hipEventSynchronize(stopEvent); 

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, startEvent, stopEvent);
	cout << milliseconds << " (ms)" << endl;

	//d2h_print1d(d_C, C, rows);
	if (check(d_C, C, rows, cols))	{
		printf("success!\n");
	}


	// release
	if (A != NULL)				checkCudaErrors(hipHostFree(A));
	if (B != NULL)				checkCudaErrors(hipHostFree(B));
	if (C != NULL)				checkCudaErrors(hipHostFree(C));

	if (d_A != NULL)			checkCudaErrors(hipFree(d_A));
	if (d_B != NULL) 			checkCudaErrors(hipFree(d_B));
	if (d_C != NULL)			checkCudaErrors(hipFree(d_C));

	hipDeviceReset();
}



int main(int argc, char **argv) {

	hipDeviceProp_t prop;
	checkCudaErrors( hipGetDeviceProperties(&prop, 0) );
	printf("Device: %s\n", prop.name);

	int rows = atoi(argv[1]);
	int cols = atoi(argv[2]);

	// 100 x 100
	for(int i=0; i<10; i++)
		test_v1a(rows,   cols);

	test_v1a(rows,   cols);

    return(0);
}
