#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel()
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	printf("cuda thread %d\n", i);
}

int main(void)
{
	// set device
	int device = 0;
	hipSetDevice(device);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device);
	printf("device %d : %s\n", device, prop.name);	

    hipError_t err = hipSuccess;

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 64;
    int blocksPerGrid = 1; 
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

	// Create cuda events
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
    kernel<<<blocksPerGrid, threadsPerBlock>>>();
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Kernel run time : %f ms\n" , milliseconds);


    err = hipDeviceReset();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

