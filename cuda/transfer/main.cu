/*
https://devblogs.nvidia.com/parallelforall/how-optimize-data-transfers-cuda-cc/

*/
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define ITERS  10

inline
hipError_t checkCuda(hipError_t result)
{
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %sn", 
				hipGetErrorString(result));
		assert(result == hipSuccess);
	}
	return result;
}

void run_pageable(unsigned int nElements)
{
	const unsigned int bytes = nElements * sizeof(float);
	printf("\nTransfer %u floats, size %f (MiB)\n", nElements, 
			bytes / (1024.f * 1024.f));

	float *h_a = (float*)malloc(bytes);
	float *h_b = (float*)malloc(bytes);
	float *d_a;
	hipMalloc((int**)&d_a, bytes);

	for (int i = 0; i < nElements; ++i) h_a[i] = static_cast<float>(i);
	memset(h_b, 0, bytes);

	float time;
	hipEvent_t startEvent, stopEvent; 
	checkCuda( hipEventCreate(&startEvent) );
	checkCuda( hipEventCreate(&stopEvent) );

	// host to device
	checkCuda( hipEventRecord(startEvent, 0) );
	for (int i = 0; i < ITERS; i++)	
		checkCuda( hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice) );
	checkCuda( hipEventRecord(stopEvent, 0) );
	checkCuda( hipEventSynchronize(stopEvent) );
	checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
	printf("  Host to Device bandwidth (GB/s): %f\n", 
			bytes * 1e-6 * (float)(ITERS) / time);

	checkCuda(hipDeviceSynchronize());

	// device to host
	checkCuda( hipEventRecord(startEvent, 0) );
	for (int i = 0; i < ITERS; i++)	
		checkCuda( hipMemcpy(h_b, d_a, bytes, hipMemcpyDeviceToHost) );
	checkCuda( hipEventRecord(stopEvent, 0) );
	checkCuda( hipEventSynchronize(stopEvent) );
	checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
	printf("  Device to Host bandwidth (GB/s): %f\n", 
			bytes * 1e-6 * (float)(ITERS) / time);

	checkCuda(hipDeviceSynchronize());

	for (int i = 0; i < nElements; ++i) {
		if (h_a[i] != h_b[i]) {
			printf("*** transfers failed ***");
			break;
		}
	}

	free(h_a);
	free(h_b);
	hipFree(d_a);
}

void run_pinned(unsigned int nElements)
{
	const unsigned int bytes = nElements * sizeof(float);
	printf("\nTransfer %u floats, size %f (MiB)\n", nElements, 
			bytes / (1024.f * 1024.f));

	float *h_a, *h_b;
	hipHostMalloc((void**)&h_a, bytes, hipHostMallocDefault);
	hipHostMalloc((void**)&h_b, bytes, hipHostMallocDefault);

	float *d_a;
	hipMalloc((int**)&d_a, bytes);

	for (int i = 0; i < nElements; ++i) h_a[i] = static_cast<float>(i);
	memset(h_b, 0, bytes);

	float time;
	hipEvent_t startEvent, stopEvent; 
	checkCuda( hipEventCreate(&startEvent) );
	checkCuda( hipEventCreate(&stopEvent) );

	// host to device
	checkCuda( hipEventRecord(startEvent, 0) );
	for (int i = 0; i < ITERS; i++)	
		checkCuda( hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice) );
	checkCuda( hipEventRecord(stopEvent, 0) );
	checkCuda( hipEventSynchronize(stopEvent) );
	checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
	printf("  Host to Device bandwidth (GB/s): %f\n", 
			bytes * 1e-6 * (float)(ITERS) / time);

	checkCuda(hipDeviceSynchronize());

	// device to host
	checkCuda( hipEventRecord(startEvent, 0) );
	for (int i = 0; i < ITERS; i++)	
		checkCuda( hipMemcpy(h_b, d_a, bytes, hipMemcpyDeviceToHost) );
	checkCuda( hipEventRecord(stopEvent, 0) );
	checkCuda( hipEventSynchronize(stopEvent) );
	checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
	printf("  Device to Host bandwidth (GB/s): %f\n", 
			bytes * 1e-6 * (float)(ITERS) / time);

	for (int i = 0; i < nElements; ++i) {
		if (h_a[i] != h_b[i]) {
			printf("*** transfers failed ***");
			break;
		}
	}

	checkCuda(hipDeviceSynchronize());

	hipHostFree(h_a);
	hipHostFree(h_b);
	hipFree(d_a);
}

void run_um(unsigned int nElements)
{
	const unsigned int bytes = nElements * sizeof(float);
	printf("\nTransfer %u floats, size %f (MiB)\n", nElements, 
			bytes / (1024.f * 1024.f));

	float *h_a, *h_b;
	hipMallocManaged((void**)&h_a, bytes);
	hipMallocManaged((void**)&h_b, bytes);

	float *d_a;
	hipMalloc((int**)&d_a, bytes);

	for (int i = 0; i < nElements; ++i) h_a[i] = static_cast<float>(i);
	memset(h_b, 0, bytes);

	float time;
	hipEvent_t startEvent, stopEvent; 
	checkCuda( hipEventCreate(&startEvent) );
	checkCuda( hipEventCreate(&stopEvent) );

	// host to device
	checkCuda( hipEventRecord(startEvent, 0) );
	for (int i = 0; i < ITERS; i++)	
		checkCuda( hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice) );
	checkCuda( hipEventRecord(stopEvent, 0) );
	checkCuda( hipEventSynchronize(stopEvent) );
	checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
	printf("  Host(um) to Device bandwidth (GB/s): %f\n", 
			bytes * 1e-6 * (float)(ITERS) / time);

	checkCuda(hipDeviceSynchronize());

	// device to host ( um )
	checkCuda( hipEventRecord(startEvent, 0) );
	for (int i = 0; i < ITERS; i++)	
		checkCuda( hipMemcpy(h_b, d_a, bytes, hipMemcpyDeviceToHost) );
	checkCuda( hipEventRecord(stopEvent, 0) );
	checkCuda( hipEventSynchronize(stopEvent) );
	checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
	printf("  Device to Host(um) bandwidth (GB/s): %f\n", 
			bytes * 1e-6 * (float)(ITERS) / time);

	checkCuda(hipDeviceSynchronize());

	for (int i = 0; i < nElements; ++i) {
		if (h_a[i] != h_b[i]) {
			printf("*** transfers failed ***");
			break;
		}
	}


	// device to device ( um  )
	checkCuda( hipEventRecord(startEvent, 0) );
	for (int i = 0; i < ITERS; i++)	
		checkCuda( hipMemcpy(h_b, d_a, bytes, hipMemcpyDeviceToDevice) );
	checkCuda( hipEventRecord(stopEvent, 0) );
	checkCuda( hipEventSynchronize(stopEvent) );
	checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
	printf("  Device to Device (um) bandwidth (GB/s): %f\n", 
			bytes * 1e-6 * (float)(ITERS) / time);

	checkCuda(hipDeviceSynchronize());

	// host (um) to host (um)
	checkCuda( hipEventRecord(startEvent, 0) );
	for (int i = 0; i < ITERS; i++)	
		checkCuda( hipMemcpy(h_b, h_a, bytes, hipMemcpyHostToHost) );
	checkCuda( hipEventRecord(stopEvent, 0) );
	checkCuda( hipEventSynchronize(stopEvent) );
	checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
	printf("  Host (um) to Host (um) bandwidth (GB/s): %f\n", 
			bytes * 1e-6 * (float)(ITERS) / time);

	checkCuda(hipDeviceSynchronize());


	// device (um) to device (um)
	checkCuda( hipEventRecord(startEvent, 0) );
	for (int i = 0; i < ITERS; i++)	
		checkCuda( hipMemcpy(h_b, h_a, bytes, hipMemcpyDeviceToDevice) );
	checkCuda( hipEventRecord(stopEvent, 0) );
	checkCuda( hipEventSynchronize(stopEvent) );
	checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
	printf("  Device (um) to Device (um) bandwidth (GB/s): %f\n", 
			bytes * 1e-6 * (float)(ITERS) / time);

	checkCuda(hipDeviceSynchronize());



	hipFree(h_a);
	hipFree(h_b);
	hipFree(d_a);
}

int main() {

	// output device info and transfer size
	hipDeviceProp_t prop;
	checkCuda( hipGetDeviceProperties(&prop, 0) );
	printf("Device: %s\n", prop.name);

	unsigned int test_size[6] = {1e3,1e4,1e5,1e6,1e7,1e8};
	printf("test cases :\n");
	for(int i=0; i<6; i++)
		printf("%u\t",test_size[i]);
	printf("\n");
	
	printf("\n-------------\n pageable memory\n-------------\n");
	for(int i=0; i<6; i++)
		run_pageable(test_size[i]);	

	printf("\n-------------\n pinned memory\n-------------\n");
	for(int i=0; i<6; i++)
		run_pinned(test_size[i]);	

	printf("\n-------------\n unified memory\n-------------\n");
	for(int i=0; i<6; i++)
		run_um(test_size[i]);	

	hipDeviceReset();

	return 0;
}
