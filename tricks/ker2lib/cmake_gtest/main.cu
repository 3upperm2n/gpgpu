#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include "kernel.h"


int main()
{
	myk<<<1,1>>>();
	printf("CUDA status: %d\n", hipDeviceSynchronize());

	return 0;
}
