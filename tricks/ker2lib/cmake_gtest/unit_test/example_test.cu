#include <gtest/gtest.h>

#include <hip/hip_runtime.h>
#include "kernel.h"

TEST(ExampleTest, comparetest) 
{
	myk<<<1,1>>>();
	printf("CUDA status: %d\n", hipDeviceSynchronize());

	int expect = 1; 
    EXPECT_EQ(1, expect);
}

